#include <iostream>
#include <hip/hip_runtime.h>
//cuda programing
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int arraySize = 5;
    const int arrayBytes = arraySize * sizeof(int);

    int h_a[arraySize] = {1, 2, 3, 4, 5};
    int h_b[arraySize] = {10, 20, 30, 40, 50};
    int h_c[arraySize];

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, arrayBytes);
    hipMalloc((void**)&d_b, arrayBytes);
    hipMalloc((void**)&d_c, arrayBytes);

    hipMemcpy(d_a, h_a, arrayBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arrayBytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(d_a, d_b, d_c, arraySize);

    hipMemcpy(h_c, d_c, arrayBytes, hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}